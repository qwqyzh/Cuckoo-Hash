//task 1

#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <cstdint>
#include <climits>
#include <chrono>
#include <random>
#include <hiprand/hiprand_kernel.h>
static const uint32_t prime32_1 = 2654435761U;
static const uint32_t prime32_2 = 2246822519U;
static const uint32_t prime32_3 = 3266489917U;
static const uint32_t prime32_4 = 668265263U;
__managed__ bool legal = true;
__host__ __device__ uint32_t Hash(uint32_t key, uint32_t seed,uint32_t size) {
    uint32_t hash = seed + prime32_4;
    hash += key * prime32_2;
    hash = (hash << 13) | (hash >> (32 - 13));
    hash *= prime32_1;
    hash = (hash ^ (hash >> 15)) * prime32_2;
    hash = (hash ^ (hash >> 13)) * prime32_3;
    hash = hash ^ (hash >> 16);
    return hash%size;
}
__global__ void insert_2_kernel (uint32_t *table, uint32_t table_size, uint32_t *keys, uint32_t key_size, int MAX_step, uint32_t seed1, uint32_t seed2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= key_size) return;
    uint32_t key = keys[idx];
    uint32_t h1 = Hash(key,seed1,table_size), h2 = Hash(key,seed2,table_size);
    if (table[h1] == key || table[h2] == key) return;
    key = atomicExch(&table[h1], key);
    uint32_t evict = h1;
    if (key == UINT_MAX) return;
    for(int i = 0; i < MAX_step; i++){
        h1 = Hash(key,seed1,table_size), h2 = Hash(key,seed2,table_size);
        if (table[h1] == key || table[h2] == key) return;
        if(evict == h1) evict = h2;
        else evict = h1;
        key = atomicExch(&table[evict], key);
        if (key == UINT_MAX) return;
    }
    legal = false;
}
__global__ void insert_3_kernel (uint32_t *table, uint32_t table_size, uint32_t *keys, uint32_t key_size, int MAX_step, uint32_t seed1, uint32_t seed2,uint32_t seed3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= key_size) return;
    uint32_t key = keys[idx];
    uint32_t h1 = Hash(key,seed1,table_size), h2 = Hash(key,seed2,table_size), h3 = Hash(key,seed3,table_size);
    if (table[h1] == key || table[h2] == key || table[h3] == key) return;
    uint32_t evict = h1;
    key = atomicExch(&table[h1], key);
    if (key == UINT_MAX) return;
    for(int i = 0; i < MAX_step; i++){
        h1 = Hash(key,seed1,table_size), h2 = Hash(key,seed2,table_size), h3 = Hash(key,seed3,table_size);
        if (table[h1] == key || table[h2] == key || table[h3] == key) return;
        if(evict == h1) evict = h2;
        else if(evict == h2) evict = h3;
        else evict = h1;
        key = atomicExch(&table[evict], key);
        if (key == UINT_MAX) return;
    } 
    legal = false;
}
__global__ void generate_key_kernel(uint32_t *keys, uint32_t key_size, uint32_t seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= key_size) return;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    keys[idx] = hiprand(&state);
}
//const uint32_t MAX_table_size = 1<<25;
//const uint32_t MAX_key_size = 1<<24;
int main() {
    int version = 0;
    hipRuntimeGetVersion(&version);
    std::cout<<"CUDA Runtime Version: " << version / 1000 << "." << (version % 1000) / 10 << std::endl;
    uint32_t MAX_step = 96;
    // int *h_table = new int[MAX_table_size];
    // int *h_key = new int[MAX_key_size];
    uint32_t key_size = 1<<24;
    int table_size = 1.4 * key_size;
    int block_size = 512;
    int grid_size = (key_size + block_size - 1) / block_size;
    srand(time(NULL));
    uint32_t *table, *keys;
    printf("2 hash functions\n");
    for(int i=0;i<10;i++){
        MAX_step = 6*(i+1);
        printf("MAX_step=%d ",MAX_step);
        grid_size = (key_size + block_size - 1) / block_size;
        hipError_t err;
        err=hipMallocManaged((void**)&table,  sizeof(uint32_t)*table_size);
        // cudaMemcpy(table, h_table, sizeof(uint32_t)*table_size, cudaMemcpyHostToDevice);
        if (err != hipSuccess) {
            printf("CUDA malloc failed: %s\n",hipGetErrorString(err)) ;
            return -1;
        }   
        // cudaMemcpy(keys, h_key, sizeof(uint32_t)*key_size, cudaMemcpyHostToDevice);
//        legal = false;
        err=hipMallocManaged((void**)&keys,  sizeof(uint32_t)*key_size);
        legal = true;
        if (err != hipSuccess) {
            printf("CUDA malloc failed for keys\n");
            return -1;
        }
        hipMemset(table,0xff,table_size * sizeof(uint32_t));
        uint32_t seed = rand(), seed1 = rand(), seed2 = rand();
        generate_key_kernel<<<grid_size, block_size>>>(keys, key_size, seed);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("CUDA synchronization error: %s\n", hipGetErrorString(err));
            return -1;
        }
        auto time1 = std::chrono::high_resolution_clock::now();
        insert_2_kernel<<<grid_size, block_size>>>(table, table_size, keys, key_size, MAX_step, seed1, seed2);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("CUDA synchronization error: %s\n", hipGetErrorString(err));
            return -1;
        }
        auto time2 = std::chrono::high_resolution_clock::now();
        long long time = std::chrono::duration_cast<std::chrono::microseconds>(time2 - time1).count();
        if(legal) printf("all keys are inserted ");
        else printf("some keys are not inserted ");
        printf("key_size=%d time=%lld MOPS=%lf\n", key_size , time, 1.0 * key_size / time);
        hipFree(table);
        hipFree(keys);
    }
    printf("3 hash functions\n");
    for(int i=0;i<10;i++){
        MAX_step = 12*(i+1);
        printf("MAX_step=%d ",MAX_step);
        grid_size = (key_size + block_size - 1) / block_size;
        hipError_t err;
        err=hipMallocManaged((void**)&table,  sizeof(uint32_t)*table_size);
        // cudaMemcpy(table, h_table, sizeof(uint32_t)*table_size, cudaMemcpyHostToDevice);
        if (err != hipSuccess) {
            printf("CUDA malloc failed: %s\n",hipGetErrorString(err)) ;
            return -1;
        }   
        // cudaMemcpy(keys, h_key, sizeof(uint32_t)*key_size, cudaMemcpyHostToDevice);
//       legal = false;
//       while(!legal){
            legal = true;
            err=hipMallocManaged((void**)&keys,  sizeof(uint32_t)*key_size);
            if (err != hipSuccess) {
                printf("CUDA malloc failed for keys\n");
                return -1;
            }
            hipMemset(table,0xff,table_size * sizeof(uint32_t));
            uint32_t seed = rand(), seed1 = rand(), seed2 = rand(), seed3 = rand();
            generate_key_kernel<<<grid_size, block_size>>>(keys, key_size, seed);
            err = hipDeviceSynchronize();
            if (err != hipSuccess) {
                printf("CUDA synchronization error: %s\n", hipGetErrorString(err));
                return -1;
            }
            auto time1 = std::chrono::high_resolution_clock::now();
            insert_3_kernel<<<grid_size, block_size>>>(table, table_size, keys, key_size, MAX_step, seed1, seed2, seed3);
            err = hipDeviceSynchronize();
            if (err != hipSuccess) {
                printf("CUDA synchronization error: %s\n", hipGetErrorString(err));
                return -1;
            }
//       }
        if(legal) printf("all keys are inserted ");
        else printf("some keys are not inserted ");
        auto time2 = std::chrono::high_resolution_clock::now();
        long long time = std::chrono::duration_cast<std::chrono::microseconds>(time2 - time1).count();
        printf("key_size=%d time=%lld MOPS: %lf\n", key_size, time, 1.0 * key_size / time);
        hipFree(table);
        hipFree(keys);
    }
    return 0;
}